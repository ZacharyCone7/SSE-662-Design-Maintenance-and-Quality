#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <>
#include <chrono>

using namespace std;

// Constants for shared memory size and array size
#define SHMEM_SIZE 256 * 4  // Shared memory size for the reduction kernel
#define SIZE 256            // Size of each block (number of threads per block)

// Function to check for CUDA errors and print error messages
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "Error: " << msg << " - " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

// xorshift128+ generator for random numbers
int xorshift128plus(uint64_t s[2]) {
    uint64_t x = s[0];
    uint64_t y = s[1];
    s[0] = y;
    x ^= x << 23;
    s[1] = x ^ y ^ (x >> 17) ^ (y >> 26);
    uint64_t result = s[1] + y;

    // Cast to int (ensure it fits in the int range)
    return static_cast<int>(result & 0x7FFFFFFF); // Mask to ensure positive 32-bit int
}

// Generate random array using xorshift128+
vector<int> generateRandomArray(uint64_t s[2], size_t size) {
    vector<int> random_array(size);
    for (size_t i = 0; i < size; i++) {
        random_array[i] = xorshift128plus(s); // Generate random number for each element
    }
    return random_array;
}

// Define the kernel for parallel reduction
__global__ void reduceKernel(int* inputArray, int* outputArray, int arraySize) {
    extern __shared__ int shared_sum[];  // Shared memory to hold partial sums

    // Calculate thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    int stride = blockDim.x * gridDim.x; // Stride: the step size by which a thread advances through the data

    // Initialize partial sum for each thread
    int partial_sum = 0;
    // Perform reduction across threads
    for (int i = tid; i < arraySize; i += stride) {
        partial_sum += inputArray[i];  // Accumulate data into partial sum
    }

    // Store partial sum in shared memory
    shared_sum[threadIdx.x] = partial_sum;
    // Synchronize threads within the block before proceeding
    __syncthreads();

    // Perform block-level reduction: combine the partial sums in shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + i]; // Combine results from threads
        }
        // Synchronize threads again before the next reduction step
        __syncthreads();
    }

    // Write the final result from the block (first thread in the block) to global memory
    if (threadIdx.x == 0) {
        outputArray[blockIdx.x] = shared_sum[0];
    }
}

// Initialize a vector with a specific value (for debugging or testing)
void initialize_vector(vector<int>& v, int n) {
    for (size_t i = 0; i < v.size(); i++) {
        v[i] = n;  // Set each element to the value n
    }
}

// Launch the reduction kernel with the given block size and grid size
void launchReduceKernel(int* dev_array, int* dev_output, int array_size, int blockSize, int numBlocks) {
    reduceKernel << <numBlocks, blockSize, blockSize * sizeof(int) >> > (dev_array, dev_output, array_size);  // Launch the kernel
}

// Memory management for device allocation and data transfer
void allocateAndCopyMemory(int** dev_array, int** dev_result, const vector<int>& random_array, size_t array_size) {
    hipError_t err = hipMalloc(dev_array, array_size * sizeof(int));
    checkCudaError(err, "hipMalloc failed!");

    err = hipMalloc(dev_result, sizeof(int) * ((array_size + 255) / 256));
    checkCudaError(err, "hipMalloc failed!");

    err = hipMemcpy(*dev_array, random_array.data(), array_size * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError(err, "hipMemcpy failed!");
}

// Cleanup CUDA resources
void cleanup(int* dev_array, int* dev_result) {
    hipFree(dev_array);
    hipFree(dev_result);
}

// Timing and result verification
void timeAndVerifyResults(int* dev_result, const vector<int>& random_array, int numBlocks, int array_size) {
    vector<int> host_result(numBlocks);
    hipMemcpy(host_result.data(), dev_result, sizeof(int) * numBlocks, hipMemcpyDeviceToHost);

    int gpu_result = 0;
    for (const auto& block_sum : host_result) {
        gpu_result += block_sum;
    }

    int cpu_result = 0;
    auto cpu_start = chrono::high_resolution_clock::now();
    for (const auto& num : random_array) {
        cpu_result += num;
    }
    auto cpu_end = chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> cpu_elapsed = cpu_end - cpu_start;

    if (gpu_result == cpu_result) {
        cout << "Results match! Sum: " << gpu_result << endl;
    }
    else {
        cout << "Results do not match! GPU Sum: " << gpu_result << ", CPU Sum: " << cpu_result << endl;
    }
    cout << "CPU Execution Time: " << cpu_elapsed.count() << " ms" << endl;
}

int main() {
    uint64_t state[2] = { 123456789, 987654321 };  // Seed for the random number generator
    size_t array_size = 4096;
    vector<int> random_array = generateRandomArray(state, array_size);

    int* dev_array;
    int* dev_result;
    allocateAndCopyMemory(&dev_array, &dev_result, random_array, array_size);

    float elapsedTime = 0.0f;
    //-------------- Test Case 1: Fixed Block Size, Varying Grid Size
    int blockSize = 256;
    for (int numBlocks = 1; numBlocks <= (array_size + 255) / 256; numBlocks *= 2) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        launchReduceKernel(dev_array, dev_result, array_size, blockSize, numBlocks);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        cout << "Grid Size: " << numBlocks << ", Block Size: " << blockSize << ", Execution Time: " << elapsedTime << " ms" << endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        timeAndVerifyResults(dev_result, random_array, numBlocks, array_size);
    }

    //-------------- Test Case 2: Fixed Grid Size, Varying Block Size
    int numBlocks = (array_size + 1024 - 1) / 1024;
    for (int blockSize = 128; blockSize <= 1024; blockSize *= 2) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        launchReduceKernel(dev_array, dev_result, array_size, 256, numBlocks);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        cout << "Grid Size: " << numBlocks << ", Block Size: 256, Execution Time: " << elapsedTime << " ms" << endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        timeAndVerifyResults(dev_result, random_array, numBlocks, array_size);
    }

    cleanup(dev_array, dev_result);
    return 0;
}