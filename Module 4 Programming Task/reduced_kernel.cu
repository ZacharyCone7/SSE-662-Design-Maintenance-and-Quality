#include <hip/hip_runtime.h>


// Define the kernel for parallel reduction
__global__ void reduceKernel(int* inputArray, int* outputArray, int arraySize) {
    extern __shared__ int shared_sum[];  // Shared memory to hold partial sums

    // Calculate thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    int stride = blockDim.x * gridDim.x; // Stride: the step size by which a thread advances through the data

    // Initialize partial sum for each thread
    int partial_sum = 0;
    // Perform reduction across threads
    for (int i = tid; i < arraySize; i += stride) {
        partial_sum += inputArray[i];  // Accumulate data into partial sum
    }

    // Store partial sum in shared memory
    shared_sum[threadIdx.x] = partial_sum;
    // Synchronize threads within the block before proceeding
    __syncthreads();

    // Perform block-level reduction: combine the partial sums in shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + i]; // Combine results from threads
        }
        // Synchronize threads again before the next reduction step
        __syncthreads();
    }

    // Write the final result from the block (first thread in the block) to global memory
    if (threadIdx.x == 0) {
        outputArray[blockIdx.x] = shared_sum[0];
    }
}